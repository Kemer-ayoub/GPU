
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>

__global__ void matrixAdd(const int *a, const int *b, int *c, int N){
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  index += (blockIdx.y * blockDim.y + threadIdx.y) * N;

  if(index < N*N){
    c[index] = a[index] + b[index];
  }

}

int getRandomValue(int min, int max) {
    return min + rand() % (max - min + 1);
}

int main() {
  // Array size of 2^10 (65536 elements)
  constexpr int N = 1 << 10;

  std::vector<int> matrix_A(N * N);
  std::vector<int> matrix_B(N * N);
  std::vector<int> matrix_C(N * N);

  for(int i = 0; i < N * N; i++) {
      matrix_A[i] = getRandomValue(1, 100);
      matrix_B[i] = getRandomValue(1, 100);
  }

  int *d_a, *d_b, *d_c;
  hipMalloc((void **)&d_a, sizeof(int)*N*N);
  hipMalloc((void **)&d_b, sizeof(int)*N*N);
  hipMalloc((void **)&d_c, sizeof(int)*N*N);

  hipMemcpy(d_a, matrix_A.data(), sizeof(int)*N*N, hipMemcpyHostToDevice);
  hipMemcpy(d_b, matrix_B.data(), sizeof(int)*N*N, hipMemcpyHostToDevice);

  dim3 dimBlock(32,16);
  dim3 dimGrid(ceil(N/32),ceil(N/16));


  matrixAdd<<<dimGrid,dimBlock>>>(d_a,d_b,d_c,N);

  hipMemcpy(matrix_C.data(), d_c, sizeof(int)*N*N, hipMemcpyDeviceToHost);


  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  std::cout << "COMPLETED SUCCESSFULLY\n";

  return 0;
}